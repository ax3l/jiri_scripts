#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>

#ifdef USE_NVML
#include <nvml.h>
#define NVML_CALL( call )				\
{										\
	nvmlReturn_t nvmlError = call;		\
	if (NVML_SUCCESS != nvmlError )	{	\
		fprintf (stderr, "NVML_ERROR: %s (%d) in %d line of %s\n", nvmlErrorString( nvmlError ), nvmlError , __LINE__, __FILE__ ); \
	}									\
}
#else
#define NVML_CALL( call )
#endif

/**
 * getNvmlDevice determines the NVML Device Id of the currently active CUDA device
 *
 * @param[out]  nvmlDeviceId    the NVML Device Id of the currently active CUDA device
 * @return                      NVML_SUCCESS in case of success. Error code of NVML API
 *                              or NVML_ERROR_UNKNOWN if CUDA Runtime API failed otherwise
 */
inline nvmlReturn_t getNvmlDevice( nvmlDevice_t* nvmlDeviceId )
{
	int activeCUDAdevice = 0;
	hipError_t hipError_t = hipGetDevice ( &activeCUDAdevice );
	if ( hipSuccess  != hipError_t )
		return NVML_ERROR_UNKNOWN;
	
	hipDeviceProp_t activeCUDAdeviceProp;
	hipError_t = hipGetDeviceProperties ( &activeCUDAdeviceProp, activeCUDAdevice );
	if ( hipSuccess  != hipError_t )
		return NVML_ERROR_UNKNOWN;
	
	unsigned int nvmlDeviceCount = 0;
	nvmlReturn_t nvmlError = nvmlDeviceGetCount ( &nvmlDeviceCount );
	if ( NVML_SUCCESS != nvmlError )
		return nvmlError;
	
	for ( unsigned int nvmlDeviceIdx = 0; nvmlDeviceIdx < nvmlDeviceCount; ++nvmlDeviceIdx )
	{
		nvmlError = nvmlDeviceGetHandleByIndex ( nvmlDeviceIdx, nvmlDeviceId );
		if ( NVML_SUCCESS != nvmlError )
			return nvmlError; 
		nvmlPciInfo_t nvmPCIInfo;
		nvmlError = nvmlDeviceGetPciInfo ( *nvmlDeviceId, &nvmPCIInfo );
		if ( NVML_SUCCESS != nvmlError )
			return nvmlError;
		//TODO: Is it sufficient to check the below? What about multi GPU boards?
		//      Do we need to consider MultiGpuBoard, multiGpuBoardGroupID of hipDeviceProp_t?
		if ( static_cast<unsigned int>(activeCUDAdeviceProp.pciBusID) == nvmPCIInfo.bus &&
		     static_cast<unsigned int>(activeCUDAdeviceProp.pciDeviceID) == nvmPCIInfo.device &&
			 static_cast<unsigned int>(activeCUDAdeviceProp.pciDomainID) == nvmPCIInfo.domain )
			break;
	}
	return NVML_SUCCESS;
}

inline nvmlReturn_t reportApplicationClocks( nvmlDevice_t nvmlDeviceId )
{
	unsigned int appSMclock = 0;
	unsigned int appMemclock = 0;
	nvmlReturn_t nvmlError = nvmlDeviceGetApplicationsClock ( nvmlDeviceId, NVML_CLOCK_SM, &appSMclock );
	if ( NVML_SUCCESS != nvmlError )
			return nvmlError;
	nvmlError = nvmlDeviceGetApplicationsClock ( nvmlDeviceId, NVML_CLOCK_MEM, &appMemclock );
	if ( NVML_SUCCESS != nvmlError )
			return nvmlError;
	
	std::cout<<"Application Clocks = ("<<appMemclock<<","<<appSMclock<<")"<<std::endl;
	return NVML_SUCCESS;
}

__global__ void test_kernel()
{
	printf("test_kernel()\n");
}

int matrixMultiply(dim3 &dimsA, dim3 &dimsB);

int main()
{
	hipSetDevice(0);
	test_kernel<<<1,1>>>();
	hipDeviceSynchronize();
	
	NVML_CALL( nvmlInit() );
	
	nvmlDevice_t nvmlDeviceId;
	NVML_CALL( getNvmlDevice( &nvmlDeviceId ) );
	
	NVML_CALL( reportApplicationClocks( nvmlDeviceId ) );
	
	unsigned int memClock = 0;
	NVML_CALL( nvmlDeviceGetClockInfo( nvmlDeviceId, NVML_CLOCK_MEM, &memClock ) );
	
	unsigned int numSupportedSMClocks = 32;
	unsigned int smClocksMHz[32];
	NVML_CALL( nvmlDeviceGetSupportedGraphicsClocks ( nvmlDeviceId, memClock, &numSupportedSMClocks, smClocksMHz ) );
	
	unsigned int numSupportedMemClocks = 32;
	unsigned int memClocksMHz[32];
	NVML_CALL( nvmlDeviceGetSupportedMemoryClocks ( nvmlDeviceId, &numSupportedMemClocks, memClocksMHz ) ); 

	unsigned int maxSMclock = 0;
	unsigned int maxMemclock = 0;
	NVML_CALL( nvmlDeviceGetMaxClockInfo ( nvmlDeviceId, NVML_CLOCK_SM, &maxSMclock ) );
	NVML_CALL( nvmlDeviceGetMaxClockInfo ( nvmlDeviceId, NVML_CLOCK_MEM, &maxMemclock ) );

	//Check permissions to modify application clocks
	nvmlEnableState_t isRestricted;
	NVML_CALL( nvmlDeviceGetAPIRestriction ( nvmlDeviceId, NVML_RESTRICTED_API_SET_APPLICATION_CLOCKS, &isRestricted ) );
	
	if ( NVML_FEATURE_DISABLED == isRestricted )
	{
		dim3 dimsA(256,256);
		dim3 dimsB(256,256);
		std::cout<<"Setting application SM clocks min value."<<std::endl;
		NVML_CALL( nvmlDeviceSetApplicationsClocks ( nvmlDeviceId, memClocksMHz[0], smClocksMHz[numSupportedSMClocks-1] ) );

		NVML_CALL( reportApplicationClocks( nvmlDeviceId ) );
		
		//Is this safe?
		matrixMultiply(dimsA, dimsB);

		std::cout<<"Setting application clocks max value."<<std::endl;
		NVML_CALL( nvmlDeviceSetApplicationsClocks ( nvmlDeviceId, maxMemclock, maxSMclock ) );
		
		NVML_CALL( reportApplicationClocks( nvmlDeviceId ) );
		matrixMultiply(dimsA, dimsB);
	}
	
	//Reset Application Clocks and Shutdown NVML
	if ( NVML_FEATURE_DISABLED == isRestricted )
	{
		NVML_CALL( nvmlDeviceResetApplicationsClocks ( nvmlDeviceId ) ); 
	}
	NVML_CALL( nvmlShutdown() );
	
	hipDeviceReset();
	return 0;
}
